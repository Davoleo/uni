/*
heat_gpu.cu

nvcc -O2 heat_gpu.cu -o heat_gpu

./heat_gpu -h
./heat_gpu -r 4096 -c 4096
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h> //gettimeofday

#ifndef BLOCK_SIZE_X
    #define BLOCK_SIZE_X 16
#endif
#ifndef BLOCK_SIZE_Y
    #define BLOCK_SIZE_Y 16 
#endif

__host__ __device__ int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__global__ void Jacobi_Iterator_GPU(const float *__restrict__ T_old, float *__restrict__ T_new, const int NX, const int NY);
__global__ void Jacobi_Iterator_GPU_Shared(const float *__restrict__ T_old, float *__restrict__ T_new, const int NX, const int NY);
__global__ void copy_constant(float *__restrict__ T, const float *__restrict__ T_const, const int NX, const int NY);

void options(int argc, char *argv[]);
void usage(char *argv[]);
void Jacobi_Iterator_CPU(float *__restrict T, float *__restrict T_new, const int NX, const int NY);
void Init_center(float *__restrict h_T, const int NX, const int NY); // center
void Init_left(float *__restrict h_T, const int NX, const int NY);   // left border
void Init_top(float *__restrict h_T, const int NX, const int NY);    // top border
void copy_rows(float *__restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void copy_cols(float *__restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void print_colormap(float *__restrict h_T);                          //

int NX = 256;               // --- Number of discretization points along the x axis
int NY = 256;               // --- Number of discretization points along the y axis
int MAX_ITER = 1000;        // --- Number of Jacobi iterations

/********/
/* MAIN */
/********/
int main(int argc, char **argv)
{

    int iter;

    double t1, t2;
    struct timeval tempo;

    options(argc, argv); /* optarg management */
    // fprintf(stderr, "# NX, NY, MAX_ITER, CPU_TIME, GPU_TIME");
    fprintf(stderr, "# NX=%d, NY=%d, MAX_ITER=%d, BLOCK_SIZE_X=%d, BLOCK_SIZE_Y=%d", NX, NY, MAX_ITER, BLOCK_SIZE_X, BLOCK_SIZE_Y);

    // --- CPU temperature distributions
    float *h_T = (float *)calloc(NX * NY, sizeof(float));
    Init_center(h_T, NX, NY);
    //   Init_left(h_T,    NX, NY);
    //Init_top(h_T, NX, NY);
    float *h_T_GPU_result = (float *)malloc(NX * NY * sizeof(float));
    float *temp;

    // --- GPU temperature distribution
    // Matrice in cui si scrive
    float *d_T;
    hipMalloc((void **)&d_T, NX * NY * sizeof(float));
    // matrice da cui si legge
    float *d_T_old;
    hipMalloc((void **)&d_T_old, NX * NY * sizeof(float));
    float *d_T_const;
    hipMalloc((void **)&d_T_const, NX * NY * sizeof(float));

    hipMemcpy(d_T, h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_T_const, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    //^ Si fa la copia deviceToDevice per evitare di usare il bus
    // e quindi fare la copia molto più velocemente

    // --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid(iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

    ///////////////////////////////

    gettimeofday(&tempo, 0);
    t1 = tempo.tv_sec + (tempo.tv_usec / 1000000.0); // get timer1

    float time_passed;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    for (iter = 0; iter < MAX_ITER; iter = iter + 1)
    {
        // copia aree costanti di calore
        copy_constant<<<dimGrid, dimBlock>>>(d_T, d_T_const, NX, NY);
        // Iteratore di diffusione calore
        
        #ifdef USE_SHARED_MEMORY
            Jacobi_Iterator_GPU_Shared<<<dimGrid, dimBlock>>>(d_T, d_T_old, NX, NY);
        #else
            Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T, d_T_old, NX, NY);
        #endif
        // I due kernel sono eseguiti sequenzialmente
        // [La CPU si occupa della sincronizzazione in automatico]

        // Scambio dei puntatori new e old tra di loro [scambio di buffer]
        temp = d_T;
        d_T = d_T_old;
        d_T_old = temp;
    }

    gettimeofday(&tempo, 0);
    t2 = tempo.tv_sec + (tempo.tv_usec / 1000000.0); // get timer2

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_passed, start, stop);

    // --- Copy result from device to host
    hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost);

    // CPU_TIME
    fprintf(stderr, "CPU_TIME=%.3f, ", t2 - t1);
    // GPU_TIME
    fprintf(stderr, "GPU_TIME=%.3f \n", time_passed / 1000);
    // CSV OUTPUT
    fprintf(stderr, "%d, %d, %d, %.3f, %.3f\n", NX, NY, MAX_ITER, t2 - t1, time_passed / 1000);

    print_colormap(h_T_GPU_result);

    free(h_T);
    hipFree(d_T);
    hipFree(d_T_old);

    return 0;
}

/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float *__restrict T, float *__restrict T_new, const int NX, const int NY)
{
    int i, j;

    // --- Only update "interior" (not boundary) node points
    for (j = 1; j < NY - 1; j++)
        for (i = 1; i < NX - 1; i++)
        {
            float T_E = T[(i + 1) + NX * j];
            float T_W = T[(i - 1) + NX * j];
            float T_N = T[i + NX * (j + 1)];
            float T_S = T[i + NX * (j - 1)];
            T_new[NX * j + i] = 0.25 * (T_E + T_W + T_N + T_S);
        }
}

/*************
***** copy constant
************/

__global__ void copy_constant(float *__restrict__ T, const float *__restrict__ T_const, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Per situazioni in cui si ha una matrice linearizzata
    //  se la lunghezza delle righe non è una potenza di 2
    //  è meglio aggiungere del padding per farla diventare una potenza di 2 per evitare disallineamento in memoria
    //  può causare fino a x2 di costo accesso in memoria.
    int P = i + j * NX;
    if (T_const[P] > 0)
        T[P] = T_const[P]; // copia punti a temperatura costante

    if (j == 0)
        T[NX * 0 + i] = T[NX * (NY - 2) + i]; // copia penultima riga nella prima
    if (j == (NY - 1))
        T[NX * (NY - 1) + i] = T[(NX * 1) + i]; // copia seconda riga nell'ultima riga
    if (i == 0)
        T[NX * j + 0] = T[NX * j + NX - 2]; // copia penultima colonna nella prima
    if (i == (NX - 1))
        T[NX * j + (NX - 1)] = T[(NX * j) + 1]; // copia seconda colonna nell'ultima
}

/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float *__restrict__ T_old, float *__restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    int left = i - 1;
    int right = i + 1;
    int up = j + 1;
    int down = j - 1;
    if (left < 0)
        left = 0;
    if (right >= gridDim.x * blockDim.x)
        right--;
    if (down < 0)
        down = 0;
    if (up >= gridDim.y * blockDim.y)
        up--;

    //                                                 N
    int P = i + j * NX;     // node (i,j)              |
    int N = i + up * NX;    // node (i,j+1)            |
    int S = i + down * NX;  // node (i,j-1)     W ---- P ---- E
    int E = right + j * NX; // node (i+1,j)            |
    int W = left + j * NX;  // node (i-1,j)            |
                            //                         S

    /// Update
    T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]);
}


/*******************************************************************/
/* JACOBI ITERATION FUNCTION USING SHARED MEMORY CACHE BLOCK - GPU */
/*******************************************************************/
__global__ void Jacobi_Iterator_GPU_Shared(const float *__restrict__ T_old, float *__restrict__ T_new, const int NX, const int NY)
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    const int CACHE_SIZEX = BLOCK_SIZE_X + 2;
    __shared__ float cache[(BLOCK_SIZE_Y + 2) * (CACHE_SIZEX)];

    int cacheX = threadIdx.x + 1;
    int cacheY = threadIdx.y + 1;

    // Fill cache center
    cache[cacheY * CACHE_SIZEX + cacheX] = T_old[i + j * NX];

    // Fill cache borders
    if (threadIdx.y == 0)
    {
        int row = j - 1;
        if (row < 0)
        {
            row = 0;
        }
        cache[(cacheY - 1) * CACHE_SIZEX + cacheX] = T_old[i + row * NX];
    }

    if (threadIdx.y == blockDim.y - 1)
    {
        int row = j + 1;
        if (row >= NY)
        {
            row--;
        }
        cache[(cacheY + 1) * CACHE_SIZEX + cacheX] = T_old[i + row * NX];
    }

    if (threadIdx.x == 0)
    {
        int col = i - 1;
        if (col < 0)
        {
            col = 0;
        }
        cache[cacheY * CACHE_SIZEX + cacheX - 1] = T_old[col + j * NX];
    }

    if (threadIdx.x == blockDim.x - 1)
    {
        int col = i + 1;
        if (col >= NX)
        {
            col--;
        }
        cache[cacheY * CACHE_SIZEX + cacheX + 1] = T_old[col + j * NX];
    }

    __syncthreads();

    int left = cacheX - 1;
    int right = cacheX + 1;
    int up = cacheY + 1;
    int down = cacheY - 1;

    int P = i + j * NX;
    float N = cache[up * CACHE_SIZEX + cacheX];
    float S = cache[down * CACHE_SIZEX + cacheX];
    float E = cache[cacheY * CACHE_SIZEX + right];
    float W = cache[cacheY * CACHE_SIZEX + left];

    /// Update
    T_new[P] = 0.25 * (E + W + N + S);

}

/********************************/
/* TEMPERATURE INITIALIZATION : */
/* parte centrale della griglia */
/********************************/
void Init_center(float *__restrict h_T, const int NX, const int NY)
{
    int i, j;
    int startx = NX / 2 - NX / 8;
    int endx = NX / 2 + NX / 8;
    int starty = NY / 2 - NY / 8;
    int endy = NY / 2 + NY / 8;
    //    int starty=NY/4;
    //    int endy=NY-NY/4;
    for (i = startx; i < endx; i++)
        for (j = starty; j < endy; j++)
            if (abs(i - (NX/2)) >= NX/12 && abs(j - (NX/2)) >= NX/12)
                    h_T[NX * j + i] = 1.0;
}

/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo sinistro               */
/********************************/
void Init_left(float *__restrict h_T, const int NX, const int NY)
{

    int i, j;
    int startx = 1;
    int endx = 2;
    int starty = 0;
    int endy = NY - 1;
    for (i = startx; i < endx; i++)
        for (j = starty; j < endy; j++)
            h_T[NX * j + i] = 1.0;
}

/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo alto                   */
/********************************/
void Init_top(float *__restrict h_T, const int NX, const int NY)
{

    int i;
    int startx = 0;
    int endx = NX - 1;
    for (i = startx; i < endx; i++)
        h_T[NX + i] = 1.0;
}

/********************************/
/* Periodic boundary conditions */
/* COPY BORDER: COLS            */
/********************************/
void copy_cols(float *__restrict h_T, const int NX, const int NY)
{

    int i;

    // copy cols
    for (i = 1; i < NY - 1; ++i)
    {
        h_T[NX * i + 0] = h_T[NX * i + NX - 2];
        h_T[NX * i + NX - 1] = h_T[NX * i + 1];
    }
}

/********************************/
/* Periodic boundary conditions */
/* COPY BOREDER: ROWS           */
/********************************/
void copy_rows(float *__restrict h_T, const int NX, const int NY)
{

    memcpy(&(h_T[NX * 0]), &(h_T[NX * (NY - 2)]), NX * sizeof(float));
    memcpy(&(h_T[NX * (NY - 1)]), &(h_T[NX * 1]), NX * sizeof(float));
}

/******************************************/
/* print color map                        */
/******************************************/

void print_colormap(float *__restrict h_T)
{
    int i, j;

    for (j = 1; j < NY - 1; j++)
    {
        for (i = 1; i < NX - 1; i++)
        {
            printf("%2.2f ", h_T[NX * j + i]);
        }
        printf("\n");
    }
}

/******************************************/
/* options management                     */
/******************************************/
void options(int argc, char *argv[])
{

    int i;
    while ((i = getopt(argc, argv, "c:r:s:C:Rm:h")) != -1)
    {
        switch (i)
        {
        case 'c':
            NX = strtol(optarg, NULL, 10);
            break;
        case 'r':
            NY = strtol(optarg, NULL, 10);
            break;
        case 's':
            MAX_ITER = strtol(optarg, NULL, 10);
            break;
        case 'h':
            usage(argv);
            exit(0);
        case '?':
            usage(argv);
            exit(1);
        default:
            usage(argv);
            exit(1);
        }
    }
}

/******************************************/
/* print help                             */
/******************************************/
void usage(char *argv[])
{

    printf("\n%s [-c ncols] [-r nrows] [-s nsteps] [-h]", argv[0]);
    printf("\n");
}
