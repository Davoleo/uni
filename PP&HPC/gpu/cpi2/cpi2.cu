
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

const int threadsPerBlock = 256;

__global__ void add(float *res)
{
    __shared__ float shr[threadsPerBlock];

    //// qui ogni thread scrive il proprio indice
    //// da modificare con il calcolo del proprio relativo pezzo di rettangoli
    //// utilizzare blockIdx.x, blockDim.x e gridDim.x per calcolare la propria posizione e le divisioni da gestire
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    shr[threadIdx.x] = (1.0 / (1.0 + x*x));
    

    __syncthreads();
    // for reductions, threadsPerBlock must be a power of 2 // because of the following code
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (threadIdx.x < i)
            shr[threadIdx.x] += shr[threadIdx.x + i];
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x == 0)
        res[blockIdx.x] = shr[threadIdx.x];
}

int main(void)
{

    int nblocks = 128;
    printf("hi\n");

    float *res = (float *)malloc(nblocks * sizeof(float));
    float *dev_res;
    HANDLE_ERROR(hipMalloc((void **)&dev_res, nblocks * sizeof(float)));
    printf("start\n");

    add<<<nblocks, threadsPerBlock>>>(dev_res);

    HANDLE_ERROR(hipMemcpy(res, dev_res, nblocks * sizeof(float), hipMemcpyDeviceToHost));
    printf("ok %f\n", res[0]);

    float total = 0;
    for (int i = 0; i < nblocks; i++)
    {
        printf("Block %d: %f\n", i, res[i]);
        total += res[i];
    }

    printf("Somma %f\n", total);
    //    cudaFree( dev_res );

    return 0;
}
