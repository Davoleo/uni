
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

const int threadsPerBlock = 256;

__global__ void add(float *res)
{
    __shared__ float shr[threadsPerBlock];

    //// qui ogni thread scrive il proprio indice
    //// da modificare con il calcolo del proprio relativo pezzo di rettangoli
    //// utilizzare blockIdx.x, blockDim.x e gridDim.x per calcolare la propria posizione e le divisioni da gestire
    int x = threadIdx.x + blockDim.x * blockIdx.x;

    //cpi calculation: f1
    shr[threadIdx.x] = (1.0 / (1.0 + x*x));

    //cpi calculation: f2
    //shr[threadIdx.x] = (sqrt(1-x*x));
    

    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2 // because of the following code
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (threadIdx.x < i)
            shr[threadIdx.x] += shr[threadIdx.x + i];
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x == 0)
        res[blockIdx.x] = shr[threadIdx.x];
}

int main(void)
{

    //Time 
    float time_passed;
    hipEvent_t start;
    hipEvent_t stop;

    int nblocks = 128;
    printf("hi\n");

    float *res = (float *)malloc(nblocks * sizeof(float));
    float *dev_res;
    HANDLE_ERROR(hipMalloc((void **)&dev_res, nblocks * sizeof(float)));
    
    printf("start\n");
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    add<<<nblocks, threadsPerBlock>>>(dev_res);

    puts("end");
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_passed, start, stop);
    
    HANDLE_ERROR(hipMemcpy(res, dev_res, nblocks * sizeof(float), hipMemcpyDeviceToHost));
    printf("ok %f, time passed: %f\n", res[0], time_passed);

    float total = 0;
    for (int i = 0; i < nblocks; i++)
    {
        printf("Block %d: %f\n", i, res[i]);
        total += res[i];
    }

    printf("Somma %f\n", total);
    //    cudaFree( dev_res );

    return 0;
}
